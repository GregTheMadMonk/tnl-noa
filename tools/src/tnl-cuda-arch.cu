
#include <hip/hip_runtime.h>
#include <stdio.h> 

int main() {
    int num_devices;
    if( hipGetDeviceCount( &num_devices ) == hipSuccess )
        for( int i = 0; i < num_devices; i++ )
        {
            hipDeviceProp_t prop;
            hipGetDeviceProperties( &prop, i );

            int compute_minor = prop.minor;
            // sm_21 is the only 'real' architecture that does not have 'virtual' counterpart
            if( prop.major == 2 )
                compute_minor = 0;

            if( i > 0 )
                printf(" ");
            printf( "-gencode arch=compute_%d%d,code=sm_%d%d",
                    prop.major, compute_minor, prop.major, prop.minor );
        }
    printf("\n");
}
