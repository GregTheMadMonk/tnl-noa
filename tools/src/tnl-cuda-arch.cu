
#include <hip/hip_runtime.h>
#include <stdio.h> 

int main() {
    int num_devices = 0;
    hipError_t error_id = hipGetDeviceCount( &num_devices );

    if( error_id != hipSuccess ) {
        fprintf(stderr, "hipGetDeviceCount returned error %d (%s)\n",
                (int) error_id, hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }

    for( int i = 0; i < num_devices; i++ ) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties( &prop, i );

        int compute_minor = prop.minor;
        // sm_21 is the only 'real' architecture that does not have 'virtual' counterpart
        if( prop.major == 2 )
            compute_minor = 0;

        if( i > 0 )
            printf(" ");
        printf( "-gencode arch=compute_%d%d,code=sm_%d%d",
                prop.major, compute_minor, prop.major, prop.minor );
    }
    printf("\n");
}
