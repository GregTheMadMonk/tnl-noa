
#include <hip/hip_runtime.h>
/***************************************************************************
                          tnlTestFunction_impl.cu  -  description
                             -------------------
    begin                : Sep 21, 2014
    copyright            : (C) 2014 by Tomas Oberhuber
    email                : tomas.oberhuber@fjfi.cvut.cz
 ***************************************************************************/

/* See Copyright Notice in tnl/Copyright */

#ifdef TEMPLATE_EXPLICIT_INSTANTIATION
#ifdef HAVE_CUDA

#include <TNL/functions/tnlTestFunction.h>

namespace TNL {

#ifdef INSTANTIATE_FLOAT
template class tnlTestFunction< 1, float, Devices::Cuda >;
template class tnlTestFunction< 2, float, Devices::Cuda >;
template class tnlTestFunction< 3, float, Devices::Cuda >;
#endif

template class tnlTestFunction< 1, double, Devices::Cuda >;
template class tnlTestFunction< 2, double, Devices::Cuda >;
template class tnlTestFunction< 3, double, Devices::Cuda >;

#ifdef INSTANTIATE_LONG_DOUBLE
template class tnlTestFunction< 1, long double, Devices::Cuda >;
template class tnlTestFunction< 2, long double, Devices::Cuda >;
template class tnlTestFunction< 3, long double, Devices::Cuda >;
#endif

} // namespace TNL

#endif
#endif
