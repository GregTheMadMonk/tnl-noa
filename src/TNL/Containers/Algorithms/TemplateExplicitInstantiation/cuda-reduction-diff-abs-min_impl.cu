#include "hip/hip_runtime.h"
/***************************************************************************
                          cuda-reduction-diff-abs-min_impl.cu  -  description
                             -------------------
    begin                : Jan 19, 2014
    copyright            : (C) 2014 by Tomas Oberhuber
    email                : tomas.oberhuber@fjfi.cvut.cz
 ***************************************************************************/

/* See Copyright Notice in tnl/Copyright */
 
#include <TNL/Containers/Algorithms/reduction-operations.h>
#include <TNL/Containers/Algorithms/Reduction.h>
 
namespace TNL {
namespace Containers {
namespace Algorithms {
   
#ifdef TEMPLATE_EXPLICIT_INSTANTIATION


/****
 * Diff abs min
 */

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMin< char, int > >
                                   ( tnlParallelReductionDiffAbsMin< char, int >& operation,
                                     const typename tnlParallelReductionDiffAbsMin< char, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMin< char, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMin< char, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMin< char, int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMin< int, int > >
                                   ( tnlParallelReductionDiffAbsMin< int, int >& operation,
                                     const typename tnlParallelReductionDiffAbsMin< int, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMin< int, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMin< int, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMin< int, int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMin< float, int > >
                                   ( tnlParallelReductionDiffAbsMin< float, int >& operation,
                                     const typename tnlParallelReductionDiffAbsMin< float, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMin< float, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMin< float, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMin< float, int> :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMin< double, int > >
                                   ( tnlParallelReductionDiffAbsMin< double, int>& operation,
                                     const typename tnlParallelReductionDiffAbsMin< double, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMin< double, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMin< double, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMin< double, int> :: ResultType& result );

#ifdef INSTANTIATE_LONG_DOUBLE
template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMin< long double, int > >
                                   ( tnlParallelReductionDiffAbsMin< long double, int>& operation,
                                     const typename tnlParallelReductionDiffAbsMin< long double, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMin< long double, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMin< long double, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMin< long double, int> :: ResultType& result );
#endif

#ifdef INSTANTIATE_LONG_INT
template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMin< char, long int > >
                                   ( tnlParallelReductionDiffAbsMin< char, long int >& operation,
                                     const typename tnlParallelReductionDiffAbsMin< char, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMin< char, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMin< char, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMin< char, long int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMin< int, long int > >
                                   ( tnlParallelReductionDiffAbsMin< int, long int >& operation,
                                     const typename tnlParallelReductionDiffAbsMin< int, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMin< int, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMin< int, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMin< int, long int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMin< float, long int > >
                                   ( tnlParallelReductionDiffAbsMin< float, long int >& operation,
                                     const typename tnlParallelReductionDiffAbsMin< float, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMin< float, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMin< float, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMin< float, long int> :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMin< double, long int > >
                                   ( tnlParallelReductionDiffAbsMin< double, long int>& operation,
                                     const typename tnlParallelReductionDiffAbsMin< double, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMin< double, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMin< double, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMin< double, long int> :: ResultType& result );

#ifdef INSTANTIATE_LONG_DOUBLE
template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMin< long double, long int > >
                                   ( tnlParallelReductionDiffAbsMin< long double, long int>& operation,
                                     const typename tnlParallelReductionDiffAbsMin< long double, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMin< long double, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMin< long double, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMin< long double, long int> :: ResultType& result );
#endif
#endif
#endif

} // namespace Algorithms
} // namespace Containers
} // namespace TNL