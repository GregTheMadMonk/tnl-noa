/***************************************************************************
                          Cuda.cu  -  description
                             -------------------
    begin                : Dec 22, 2014
    copyright            : (C) 2014 by Tomas Oberhuber
    email                : tomas.oberhuber@fjfi.cvut.cz
 ***************************************************************************/

/* See Copyright Notice in tnl/Copyright */

#include <TNL/Devices/Cuda.h>
#include <TNL/Config/ConfigDescription.h>
#include <TNL/Config/ParameterContainer.h>

namespace TNL {
namespace Devices {

/*void Cuda::configSetup( tnlConfigDescription& config, const String& prefix )
{
#ifdef HAVE_CUDA
   config.addEntry< int >( prefix + "cuda-device", "Choose CUDA device.", 0 );
#else
   config.addEntry< int >( prefix + "cuda-device", "Choose CUDA device (CUDA is not supported on this system).", 0 );
#endif
}
 
bool Cuda::setup( const tnlParameterContainer& parameters,
                    const String& prefix )
{
   int cudaDevice = parameters.getParameter< int >( prefix + "cuda-device" );
#ifdef HAVE_CUDA
    hipSetDevice( cudaDevice );
    checkCudaDevice;
#endif
   return true;
}
*/

bool Cuda::checkDevice( const char* file_name, int line )
{
   hipError_t error = hipGetLastError();
   if( error == hipSuccess )
      return true;
   std::cerr << "CUDA ERROR(" << error << ") at line " << line << " in " << file_name << ":" << std::endl;
   switch( error )
   {
      // 1
      case hipErrorMissingConfiguration:
         std::cerr
          << "The device function being invoked (usually via ::hipLaunchByPtr()) was not " << std::endl
          << "previously configured via the ::hipConfigureCall() function. " << std::endl;
       break;

      // 2
      case hipErrorOutOfMemory:
         std::cerr
          << "The API call failed because it was unable to allocate enough memory to " << std::endl
          << "perform the requested operation. " << std::endl;
       break;

      // 3
      case hipErrorNotInitialized:
         std::cerr
          << "The API call failed because the CUDA driver and runtime could not be " << std::endl
          << "initialized. " << std::endl;
       break;
 
      // 4
      case hipErrorLaunchFailure:
         std::cerr
          << "An exception occurred on the device while executing a kernel. Common " << std::endl
          << "causes include dereferencing an invalid device pointer and accessing " << std::endl
          << "out of bounds shared memory. The device cannot be used until " << std::endl
          << "::hipDeviceReset() is called. All existing device memory allocations " << std::endl
          << "are invalid and must be reconstructed if the program is to continue " << std::endl
          << "using CUDA. " << std::endl;
       break;

      // 5
      case hipErrorPriorLaunchFailure:
         std::cerr
          << "This indicated that a previous kernel launch failed. This was previously " << std::endl
          << "used for device emulation of kernel launches. " << std::endl
          << "This error return is deprecated as of CUDA 3.1. Device emulation mode was " << std::endl
          << "removed with the CUDA 3.1 release. " << std::endl;
       break;

      // 6
      case hipErrorLaunchTimeOut:
         std::cerr
          << "This indicates that the device kernel took too long to execute. This can " << std::endl
          << "only occur if timeouts are enabled - see the device property " << std::endl
          << "ref ::hipDeviceProp_t::kernelExecTimeoutEnabled \"kernelExecTimeoutEnabled\" " << std::endl
          << "for more information. The device cannot be used until ::hipDeviceReset() " << std::endl
          << "is called. All existing device memory allocations are invalid and must be " << std::endl
          << "reconstructed if the program is to continue using CUDA. " << std::endl;
       break;

      // 7
      case hipErrorLaunchOutOfResources:
         std::cerr
          << "This indicates that a launch did not occur because it did not have " << std::endl
          << "appropriate resources. Although this error is similar to " << std::endl
          << "::hipErrorInvalidConfiguration, this error usually indicates that the " << std::endl
          << "user has attempted to pass too many arguments to the device kernel, or the " << std::endl
          << "kernel launch specifies too many threads for the kernel's register count. " << std::endl;
       break;

      // 8
      case hipErrorInvalidDeviceFunction:
         std::cerr
          << "The requested device function does not exist or is not compiled for the " << std::endl
          << "proper device architecture. " << std::endl;
       break;
 
      // 9
      case hipErrorInvalidConfiguration:
         std::cerr
          << "This indicates that a kernel launch is requesting resources that can " << std::endl
          << "never be satisfied by the current device. Requesting more shared memory " << std::endl
          << "per block than the device supports will trigger this error, as will " << std::endl
          << "requesting too many threads or blocks. See ::hipDeviceProp_t for more " << std::endl
          << "device limitations. " << std::endl;
       break;

      // 10
      case hipErrorInvalidDevice:
         std::cerr
          << "This indicates that the device ordinal supplied by the user does not " << std::endl
          << "correspond to a valid CUDA device. " << std::endl;
       break;

      // 11
      case hipErrorInvalidValue:
         std::cerr
          << "This indicates that one or more of the parameters passed to the API call " << std::endl
          << "is not within an acceptable range of values. " << std::endl;
       break;

      // 12
      case hipErrorInvalidPitchValue:
         std::cerr
          << "This indicates that one or more of the pitch-related parameters passed " << std::endl
          << "to the API call is not within the acceptable range for pitch. " << std::endl;
       break;

      // 13
      case hipErrorInvalidSymbol:
         std::cerr
          << "This indicates that the symbol name/identifier passed to the API call " << std::endl
          << "is not a valid name or identifier. " << std::endl;
       break;

      // 14
      case hipErrorMapFailed:
      std::cerr
       << "This indicates that the buffer object could not be mapped. " << std::endl;
       break;

      // 15
      case hipErrorUnmapFailed:
         std::cerr
          << "This indicates that the buffer object could not be unmapped. " << std::endl;
       break;

      // 16
      case cudaErrorInvalidHostPointer:
         std::cerr
          << "This indicates that at least one host pointer passed to the API call is " << std::endl
          << "not a valid host pointer. " << std::endl;
       break;

      // 17
      case hipErrorInvalidDevicePointer:
         std::cerr
          << "This indicates that at least one device pointer passed to the API call is " << std::endl
          << "not a valid device pointer. " << std::endl;
       break;

      case hipErrorInvalidTexture:
         std::cerr
          << "This indicates that the texture passed to the API call is not a valid " << std::endl
          << "texture. " << std::endl;
       break;

      case cudaErrorInvalidTextureBinding:
         std::cerr
          << "This indicates that the texture binding is not valid. This occurs if you " << std::endl
          << "call ::hipGetTextureAlignmentOffset() with an unbound texture. " << std::endl;
       break;

      case hipErrorInvalidChannelDescriptor:
         std::cerr
          << "This indicates that the channel descriptor passed to the API call is not " << std::endl
          << "valid. This occurs if the format is not one of the formats specified by " << std::endl
          << "::hipChannelFormatKind, or if one of the dimensions is invalid. " << std::endl;
       break;

      case hipErrorInvalidMemcpyDirection:
         std::cerr
          << "This indicates that the direction of the memcpy passed to the API call is " << std::endl
          << "not one of the types specified by ::hipMemcpyKind. " << std::endl;
       break;

      case cudaErrorAddressOfConstant:
         std::cerr
          << "This indicated that the user has taken the address of a constant variable, " << std::endl
          << "which was forbidden up until the CUDA 3.1 release. " << std::endl
          << "This error return is deprecated as of CUDA 3.1. Variables in constant " << std::endl
          << "memory may now have their address taken by the runtime via " << std::endl
          << "::hipGetSymbolAddress(). " << std::endl;
       break;

      case cudaErrorTextureFetchFailed:
         std::cerr
          << "This indicated that a texture fetch was not able to be performed. " << std::endl
          << "This was previously used for device emulation of texture operations. " << std::endl
          << "This error return is deprecated as of CUDA 3.1. Device emulation mode was " << std::endl
          << "removed with the CUDA 3.1 release. " << std::endl;
       break;

      case cudaErrorTextureNotBound:
         std::cerr
          << "This indicated that a texture was not bound for access. " << std::endl
          << "This was previously used for device emulation of texture operations. " << std::endl
          << "This error return is deprecated as of CUDA 3.1. Device emulation mode was " << std::endl
          << "removed with the CUDA 3.1 release. " << std::endl;
       break;

      case cudaErrorSynchronizationError:
         std::cerr
          << "This indicated that a synchronization operation had failed. " << std::endl
          << "This was previously used for some device emulation functions. " << std::endl
          << "This error return is deprecated as of CUDA 3.1. Device emulation mode was " << std::endl
          << "removed with the CUDA 3.1 release. " << std::endl;
       break;

      case cudaErrorInvalidFilterSetting:
         std::cerr
          << "This indicates that a non-float texture was being accessed with linear " << std::endl
          << "filtering. This is not supported by CUDA. " << std::endl;
       break;

      case cudaErrorInvalidNormSetting:
         std::cerr
          << "This indicates that an attempt was made to read a non-float texture as a " << std::endl
          << "normalized float. This is not supported by CUDA. " << std::endl;
       break;

      case cudaErrorMixedDeviceExecution:
         std::cerr
          << "Mixing of device and device emulation code was not allowed. " << std::endl
          << "This error return is deprecated as of CUDA 3.1. Device emulation mode was " << std::endl
          << "removed with the CUDA 3.1 release. " << std::endl;
       break;

      case hipErrorDeinitialized:
         std::cerr
          << "This indicated an issue with calling API functions during the unload " << std::endl
          << "process of the CUDA runtime in prior releases. " << std::endl
          << "This error return is deprecated as of CUDA 3.2. " << std::endl;
       break;

      case hipErrorUnknown:
         std::cerr
          << "This indicates that an unknown internal error has occurred. " << std::endl;
       break;

      case cudaErrorNotYetImplemented:
         std::cerr
          << "This indicates that the API call is not yet implemented. Production " << std::endl
          << "releases of CUDA will never return this error. " << std::endl;
       break;

      case cudaErrorMemoryValueTooLarge:
         std::cerr
          << "This indicated that an emulated device pointer exceeded the 32-bit address " << std::endl
          << "range. " << std::endl
          << "This error return is deprecated as of CUDA 3.1. Device emulation mode was " << std::endl
          << "removed with the CUDA 3.1 release. " << std::endl;
       break;

      case hipErrorInvalidHandle:
         std::cerr
          << "This indicates that a resource handle passed to the API call was not " << std::endl
          << "valid. Resource handles are opaque types like ::hipStream_t and " << std::endl
          << "::hipEvent_t. " << std::endl;
       break;

      case hipErrorNotReady:
         std::cerr
          << "This indicates that asynchronous operations issued previously have not " << std::endl
          << "completed yet. This result is not actually an error, but must be indicated " << std::endl
          << "differently than ::hipSuccess (which indicates completion). Calls that " << std::endl
          << "may return this value include ::hipEventQuery() and ::hipStreamQuery(). " << std::endl;
       break;

      case hipErrorInsufficientDriver:
         std::cerr
          << "This indicates that the installed NVIDIA CUDA driver is older than the " << std::endl
          << "CUDA runtime library. This is not a supported configuration. Users should " << std::endl
          << "install an updated NVIDIA display driver to allow the application to run. " << std::endl;
       break;

      case hipErrorSetOnActiveProcess:
         std::cerr
          << "This indicates that the user has called ::hipSetDevice(), " << std::endl
          << "::hipSetValidDevices(), ::hipSetDeviceFlags(), " << std::endl
          << "::cudaD3D9SetDirect3DDevice(), ::cudaD3D10SetDirect3DDevice, " << std::endl
          << "::cudaD3D11SetDirect3DDevice(), * or ::cudaVDPAUSetVDPAUDevice() after " << std::endl
          << "initializing the CUDA runtime by calling non-device management operations " << std::endl
          << "(allocating memory and launching kernels are examples of non-device " << std::endl
          << "management operations). This error can also be returned if using " << std::endl
          << "runtime/driver interoperability and there is an existing ::hipCtx_t " << std::endl
          << "active on the host thread. " << std::endl;
       break;

      case cudaErrorInvalidSurface:
         std::cerr
          << "This indicates that the surface passed to the API call is not a valid " << std::endl
          << "surface. " << std::endl;
       break;

      case hipErrorNoDevice:
      std::cerr
       << "This indicates that no CUDA-capable devices were detected by the installed " << std::endl
       << "CUDA driver. " << std::endl;
       break;

      case hipErrorECCNotCorrectable:
      std::cerr
       << "This indicates that an uncorrectable ECC error was detected during " << std::endl
       << "execution. " << std::endl;
       break;

      case hipErrorSharedObjectSymbolNotFound:
      std::cerr
       << "This indicates that a link to a shared object failed to resolve. " << std::endl;
       break;

      case hipErrorSharedObjectInitFailed:
      std::cerr
       << "This indicates that initialization of a shared object failed. " << std::endl;
       break;

      case hipErrorUnsupportedLimit:
      std::cerr
       << "This indicates that the ::hipLimit_t passed to the API call is not " << std::endl
       << "supported by the active device. " << std::endl;
       break;

      case cudaErrorDuplicateVariableName:
      std::cerr
       << "This indicates that multiple global or constant variables (across separate " << std::endl
       << "CUDA source files in the application) share the same string name. " << std::endl;
       break;

      case cudaErrorDuplicateTextureName:
      std::cerr
       << "This indicates that multiple textures (across separate CUDA source " << std::endl
       << "files in the application) share the same string name. " << std::endl;
       break;

      case cudaErrorDuplicateSurfaceName:
      std::cerr
       << "This indicates that multiple surfaces (across separate CUDA source " << std::endl
       << "files in the application) share the same string name. " << std::endl;
       break;

      case cudaErrorDevicesUnavailable:
      std::cerr
       << "This indicates that all CUDA devices are busy or unavailable at the current " << std::endl
       << "time. Devices are often busy/unavailable due to use of " << std::endl
       << "::hipComputeModeExclusive or ::hipComputeModeProhibited. They can also " << std::endl
       << "be unavailable due to memory constraints on a device that already has " << std::endl
       << "active CUDA work being performed. " << std::endl;
       break;

      case hipErrorInvalidImage:
      std::cerr
       << "This indicates that the device kernel image is invalid. " << std::endl;
       break;

      case hipErrorNoBinaryForGpu:
      std::cerr
       << "This indicates that there is no kernel image available that is suitable " << std::endl
       << "for the device. This can occur when a user specifies code generation " << std::endl
       << "options for a particular CUDA source file that do not include the " << std::endl
       << "corresponding device configuration. " << std::endl;
       break;

      case cudaErrorIncompatibleDriverContext:
      std::cerr
       << "This indicates that the current context is not compatible with this " << std::endl
       << "version of the CUDA Runtime. This can only occur if you are using CUDA " << std::endl
       << "Runtime/Driver interoperability and have created an existing Driver " << std::endl
       << "context using an older API. Please see \ref CUDART_DRIVER " << std::endl
       << "\"Interactions with the CUDA Driver API\" for more information. " << std::endl;
       break;

      case cudaErrorStartupFailure:
      std::cerr
       << "This indicates an internal startup failure in the CUDA runtime. " << std::endl;
       break;

      case cudaErrorApiFailureBase:
      std::cerr
       << "Any unhandled CUDA driver error is added to this value and returned via " << std::endl
       << "the runtime. Production releases of CUDA should not return such errors. " << std::endl;
       break;

   }
   //throw EXIT_FAILURE;
   return false;
}

} // namespace Devices
} // namespace TNL
