#include "hip/hip_runtime.h"
/***************************************************************************
                          Cuda.cu  -  description
                             -------------------
    begin                : Dec 22, 2014
    copyright            : (C) 2014 by Tomas Oberhuber
    email                : tomas.oberhuber@fjfi.cvut.cz
 ***************************************************************************/

/* See Copyright Notice in tnl/Copyright */

#include <TNL/Devices/Cuda.h>
#include <TNL/Exceptions/CudaRuntimeError.h>
#include <TNL/Config/ConfigDescription.h>
#include <TNL/Config/ParameterContainer.h>

namespace TNL {
namespace Devices {


void Cuda::setupThreads( const dim3& blockSize,
                         dim3& blocksCount,
                         dim3& gridsCount,
                         long long int xThreads,
                         long long int yThreads,
                         long long int zThreads )
{
   blocksCount.x = max( 1, xThreads / blockSize.x + ( xThreads % blockSize.x != 0 ) );
   blocksCount.y = max( 1, yThreads / blockSize.y + ( yThreads % blockSize.y != 0 ) );
   blocksCount.z = max( 1, zThreads / blockSize.z + ( zThreads % blockSize.z != 0 ) );
   
   /****
    * TODO: Fix the following:
    * I do not known how to get max grid size in kernels :(
    * 
    * Also, this is very slow. */
   /*int currentDevice( 0 );
   hipGetDevice( currentDevice );
   hipDeviceProp_t properties;
   hipGetDeviceProperties( &properties, currentDevice );
   gridsCount.x = blocksCount.x / properties.maxGridSize[ 0 ] + ( blocksCount.x % properties.maxGridSize[ 0 ] != 0 );
   gridsCount.y = blocksCount.y / properties.maxGridSize[ 1 ] + ( blocksCount.y % properties.maxGridSize[ 1 ] != 0 );
   gridsCount.z = blocksCount.z / properties.maxGridSize[ 2 ] + ( blocksCount.z % properties.maxGridSize[ 2 ] != 0 );
   */
   gridsCount.x = blocksCount.x / getMaxGridSize() + ( blocksCount.x % getMaxGridSize() != 0 );
   gridsCount.y = blocksCount.y / getMaxGridSize() + ( blocksCount.y % getMaxGridSize() != 0 );
   gridsCount.z = blocksCount.z / getMaxGridSize() + ( blocksCount.z % getMaxGridSize() != 0 );
}

void Cuda::setupGrid( const dim3& blocksCount,
                      const dim3& gridsCount,
                      const dim3& gridIdx,
                      dim3& gridSize )
{
   /* TODO: this is extremely slow!!!!
   int currentDevice( 0 );
   hipGetDevice( &currentDevice );
   hipDeviceProp_t properties;
   hipGetDeviceProperties( &properties, currentDevice );*/
 
   /****
    * TODO: fix the following
   if( gridIdx.x < gridsCount.x )
      gridSize.x = properties.maxGridSize[ 0 ];
   else
      gridSize.x = blocksCount.x % properties.maxGridSize[ 0 ];
   
   if( gridIdx.y < gridsCount.y )
      gridSize.y = properties.maxGridSize[ 1 ];
   else
      gridSize.y = blocksCount.y % properties.maxGridSize[ 1 ];

   if( gridIdx.z < gridsCount.z )
      gridSize.z = properties.maxGridSize[ 2 ];
   else
      gridSize.z = blocksCount.z % properties.maxGridSize[ 2 ];*/
   
   if( gridIdx.x < gridsCount.x - 1 )
      gridSize.x = getMaxGridSize();
   else
      gridSize.x = blocksCount.x % getMaxGridSize();
   
   if( gridIdx.y < gridsCount.y - 1 )
      gridSize.y = getMaxGridSize();
   else
      gridSize.y = blocksCount.y % getMaxGridSize();

   if( gridIdx.z < gridsCount.z - 1 )
      gridSize.z = getMaxGridSize();
   else
      gridSize.z = blocksCount.z % getMaxGridSize();
}

void Cuda::printThreadsSetup( const dim3& blockSize,
                              const dim3& blocksCount,
                              const dim3& gridSize,
                              const dim3& gridsCount,
                              std::ostream& str )
{
   str << "Block size: " << blockSize << std::endl
       << " Blocks count: " << blocksCount << std::endl
       << " Grid size: " << gridSize << std::endl
       << " Grids count: " << gridsCount << std::endl;
}


void Cuda::checkDevice( const char* file_name, int line, hipError_t error )
{
   if( error != hipSuccess )
      throw Exceptions::CudaRuntimeError( error, file_name, line );
}

std::ostream& operator << ( std::ostream& str, const dim3& d )
{
   str << "( " << d.x << ", " << d.y << ", " << d.z << " )";
   return str;
}

} // namespace Devices
} // namespace TNL
