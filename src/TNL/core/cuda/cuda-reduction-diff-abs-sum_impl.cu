#include "hip/hip_runtime.h"
/***************************************************************************
                          cuda-reduction-diff-abs-sum_impl.cu  -  description
                             -------------------
    begin                : Jan 19, 2014
    copyright            : (C) 2014 by Tomas Oberhuber
    email                : tomas.oberhuber@fjfi.cvut.cz
 ***************************************************************************/

/* See Copyright Notice in tnl/Copyright */
 
#include <TNL/core/cuda/reduction-operations.h>
#include <TNL/core/cuda/cuda-reduction.h>
 
namespace TNL {

#ifdef TEMPLATE_EXPLICIT_INSTANTIATION

/****
 * Diff abs sum
 */

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsSum< char, int > >
                                   ( tnlParallelReductionDiffAbsSum< char, int >& operation,
                                     const typename tnlParallelReductionDiffAbsSum< char, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsSum< char, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsSum< char, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsSum< char, int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsSum< int, int > >
                                   ( tnlParallelReductionDiffAbsSum< int, int >& operation,
                                     const typename tnlParallelReductionDiffAbsSum< int, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsSum< int, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsSum< int, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsSum< int, int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsSum< float, int > >
                                   ( tnlParallelReductionDiffAbsSum< float, int >& operation,
                                     const typename tnlParallelReductionDiffAbsSum< float, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsSum< float, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsSum< float, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsSum< float, int> :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsSum< double, int > >
                                   ( tnlParallelReductionDiffAbsSum< double, int>& operation,
                                     const typename tnlParallelReductionDiffAbsSum< double, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsSum< double, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsSum< double, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsSum< double, int> :: ResultType& result );

#ifdef INSTANTIATE_LONG_DOUBLE
template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsSum< long double, int > >
                                   ( tnlParallelReductionDiffAbsSum< long double, int>& operation,
                                     const typename tnlParallelReductionDiffAbsSum< long double, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsSum< long double, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsSum< long double, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsSum< long double, int> :: ResultType& result );
#endif

#ifdef INSTANTIATE_LONG_INT
template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsSum< char, long int > >
                                   ( tnlParallelReductionDiffAbsSum< char, long int >& operation,
                                     const typename tnlParallelReductionDiffAbsSum< char, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsSum< char, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsSum< char, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsSum< char, long int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsSum< int, long int > >
                                   ( tnlParallelReductionDiffAbsSum< int, long int >& operation,
                                     const typename tnlParallelReductionDiffAbsSum< int, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsSum< int, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsSum< int, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsSum< int, long int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsSum< float, long int > >
                                   ( tnlParallelReductionDiffAbsSum< float, long int >& operation,
                                     const typename tnlParallelReductionDiffAbsSum< float, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsSum< float, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsSum< float, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsSum< float, long int> :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsSum< double, long int > >
                                   ( tnlParallelReductionDiffAbsSum< double, long int>& operation,
                                     const typename tnlParallelReductionDiffAbsSum< double, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsSum< double, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsSum< double, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsSum< double, long int> :: ResultType& result );

#ifdef INSTANTIATE_LONG_DOUBLE
template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsSum< long double, long int > >
                                   ( tnlParallelReductionDiffAbsSum< long double, long int>& operation,
                                     const typename tnlParallelReductionDiffAbsSum< long double, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsSum< long double, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsSum< long double, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsSum< long double, long int> :: ResultType& result );
#endif
#endif
#endif

} // namespace TNL