
#include <hip/hip_runtime.h>
/***************************************************************************
                          TestFunction_impl.cu  -  description
                             -------------------
    begin                : Sep 21, 2014
    copyright            : (C) 2014 by Tomas Oberhuber
    email                : tomas.oberhuber@fjfi.cvut.cz
 ***************************************************************************/

/* See Copyright Notice in tnl/Copyright */

#ifdef TEMPLATE_EXPLICIT_INSTANTIATION
#ifdef HAVE_CUDA

#include <TNL/Functions/TestFunction.h>

namespace TNL {
namespace Functions {

#ifdef INSTANTIATE_FLOAT
template class TestFunction< 1, float, Devices::Cuda >;
template class TestFunction< 2, float, Devices::Cuda >;
template class TestFunction< 3, float, Devices::Cuda >;
#endif

template class TestFunction< 1, double, Devices::Cuda >;
template class TestFunction< 2, double, Devices::Cuda >;
template class TestFunction< 3, double, Devices::Cuda >;

#ifdef INSTANTIATE_LONG_DOUBLE
template class TestFunction< 1, long double, Devices::Cuda >;
template class TestFunction< 2, long double, Devices::Cuda >;
template class TestFunction< 3, long double, Devices::Cuda >;
#endif

} // namespace Functions
} // namespace TNL

#endif
#endif
