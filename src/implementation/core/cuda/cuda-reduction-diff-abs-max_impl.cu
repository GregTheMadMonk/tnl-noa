#include "hip/hip_runtime.h"
/***************************************************************************
                          cuda-reduction-diff-abs-max_impl.cu  -  description
                             -------------------
    begin                : Jan 19, 2014
    copyright            : (C) 2014 by Tomas Oberhuber
    email                : tomas.oberhuber@fjfi.cvut.cz
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/
 
#include <core/cuda/reduction-operations.h>
#include <core/cuda/cuda-reduction.h>
 
#ifdef TEMPLATE_EXPLICIT_INSTANTIATION

/****
 * Diff abs max
 */

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMax< char, int > >
                                   ( const tnlParallelReductionDiffAbsMax< char, int >& operation,
                                     const typename tnlParallelReductionDiffAbsMax< char, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMax< char, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMax< char, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMax< char, int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMax< int, int > >
                                   ( const tnlParallelReductionDiffAbsMax< int, int >& operation,
                                     const typename tnlParallelReductionDiffAbsMax< int, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMax< int, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMax< int, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMax< int, int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMax< float, int > >
                                   ( const tnlParallelReductionDiffAbsMax< float, int >& operation,
                                     const typename tnlParallelReductionDiffAbsMax< float, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMax< float, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMax< float, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMax< float, int> :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMax< double, int > >
                                   ( const tnlParallelReductionDiffAbsMax< double, int>& operation,
                                     const typename tnlParallelReductionDiffAbsMax< double, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMax< double, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMax< double, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMax< double, int> :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMax< long double, int > >
                                   ( const tnlParallelReductionDiffAbsMax< long double, int>& operation,
                                     const typename tnlParallelReductionDiffAbsMax< long double, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMax< long double, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMax< long double, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMax< long double, int> :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMax< char, long int > >
                                   ( const tnlParallelReductionDiffAbsMax< char, long int >& operation,
                                     const typename tnlParallelReductionDiffAbsMax< char, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMax< char, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMax< char, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMax< char, long int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMax< int, long int > >
                                   ( const tnlParallelReductionDiffAbsMax< int, long int >& operation,
                                     const typename tnlParallelReductionDiffAbsMax< int, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMax< int, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMax< int, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMax< int, long int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMax< float, long int > >
                                   ( const tnlParallelReductionDiffAbsMax< float, long int >& operation,
                                     const typename tnlParallelReductionDiffAbsMax< float, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMax< float, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMax< float, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMax< float, long int> :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMax< double, long int > >
                                   ( const tnlParallelReductionDiffAbsMax< double, long int>& operation,
                                     const typename tnlParallelReductionDiffAbsMax< double, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMax< double, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMax< double, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMax< double, long int> :: ResultType& result );

/*template bool reductionOnCudaDevice< tnlParallelReductionDiffAbsMax< long double, long int > >
                                   ( const tnlParallelReductionDiffAbsMax< long double, long int>& operation,
                                     const typename tnlParallelReductionDiffAbsMax< long double, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffAbsMax< long double, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffAbsMax< long double, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffAbsMax< long double, long int> :: ResultType& result );*/

#endif                                     