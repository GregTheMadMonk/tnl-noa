#include "hip/hip_runtime.h"
/***************************************************************************
                          tnlVectorOperations_impl.cu  -  description
                             -------------------
    begin                : Jul 20, 2013
    copyright            : (C) 2013 by Tomas Oberhuber
    email                : tomas.oberhuber@fjfi.cvut.cz
 ***************************************************************************/

/***************************************************************************
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 ***************************************************************************/

#include <core/vectors/tnlVectorOperations.h> 

#ifdef TEMPLATE_EXPLICIT_INSTANTIATION

/****
 * Max
 */
template int         tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< long int, tnlCuda, int >& v );
template float       tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< float, tnlCuda, int >& v );
template double      tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< double, tnlCuda, int >& v );
//template long double tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< long double, tnlCuda, int >& v );
template int         tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< long int, tnlCuda, long int >& v );
template float       tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< float, tnlCuda, long int >& v );
template double      tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< double, tnlCuda, long int >& v );
//template long double tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< long double, tnlCuda, long int >& v );

/****
 * Min
 */
template int         tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< long int, tnlCuda, int >& v );
template float       tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< float, tnlCuda, int >& v );
template double      tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< double, tnlCuda, int >& v );
//template long double tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< long double, tnlCuda, int >& v );
template int         tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< long int, tnlCuda, long int >& v );
template float       tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< float, tnlCuda, long int >& v );
template double      tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< double, tnlCuda, long int >& v );
//template long double tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< long double, tnlCuda, long int >& v );

/****
 * Abs max
 */
template int         tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< long int, tnlCuda, int >& v );
template float       tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< float, tnlCuda, int >& v );
template double      tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< double, tnlCuda, int >& v );
//template long double tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< long double, tnlCuda, int >& v );
template int         tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< long int, tnlCuda, long int >& v );
template float       tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< float, tnlCuda, long int >& v );
template double      tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< double, tnlCuda, long int >& v );
//template long double tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< long double, tnlCuda, long int >& v );

/****
 * Abs min
 */
template int         tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< long int, tnlCuda, int >& v );
template float       tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< float, tnlCuda, int >& v );
template double      tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< double, tnlCuda, int >& v );
//template long double tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< long double, tnlCuda, int >& v );
template int         tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< long int, tnlCuda, long int >& v );
template float       tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< float, tnlCuda, long int >& v );
template double      tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< double, tnlCuda, long int >& v );
//template long double tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< long double, tnlCuda, long int >& v );

/****
 * Lp norm
 */
template int         tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< int, tnlCuda, int >& v, const int& p );
template long int    tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< long int, tnlCuda, int >& v, const long int& p );
template float       tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< float, tnlCuda, int >& v, const float& p );
template double      tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< double, tnlCuda, int >& v, const double& p );
//template long double tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< long double, tnlCuda, int >& v, const long double& p );
template int         tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< int, tnlCuda, long int >& v, const int& p );
template long int    tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< long int, tnlCuda, long int >& v, const long int& p );
template float       tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< float, tnlCuda, long int >& v, const float& p );
template double      tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< double, tnlCuda, long int >& v, const double& p );
//template long double tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< long double, tnlCuda, long int >& v, const long double& p );

/****
 * Sum
 */
template int         tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< long int, tnlCuda, int >& v );
template float       tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< float, tnlCuda, int >& v );
template double      tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< double, tnlCuda, int >& v );
//template long double tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< long double, tnlCuda, int >& v );
template int         tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< long int, tnlCuda, long int >& v );
template float       tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< float, tnlCuda, long int >& v );
template double      tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< double, tnlCuda, long int >& v );
//template long double tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< long double, tnlCuda, long int >& v );

/****
 * Difference max
 */
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< int, tnlCuda, int >& v1, const tnlVector< int, tnlCuda, int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< long int, tnlCuda, int >& v1, const tnlVector< long int, tnlCuda, int >& v2 );
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< float, tnlCuda, int >& v1,  const tnlVector< float, tnlCuda, int >& v2);
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< double, tnlCuda, int >& v1, const tnlVector< double, tnlCuda, int >& v2 );
//template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< long double, tnlCuda, int >& v1, const tnlVector< long double, tnlCuda, int >& v2 );
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< int, tnlCuda, long int >& v1, const tnlVector< int, tnlCuda, long int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< long int, tnlCuda, long int >& v1, const tnlVector< long int, tnlCuda, long int >& v2 );
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< float, tnlCuda, long int >& v1, const tnlVector< float, tnlCuda, long int >& v2 );
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< double, tnlCuda, long int >& v1, const tnlVector< double, tnlCuda, long int >& v2 );
//template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< long double, tnlCuda, long int >& v1, const tnlVector< long double, tnlCuda, long int >& v2 );

/****
 * Difference min
 */
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< int, tnlCuda, int >& v1, const tnlVector< int, tnlCuda, int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< long int, tnlCuda, int >& v1, const tnlVector< long int, tnlCuda, int >& v2 );
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< float, tnlCuda, int >& v1,  const tnlVector< float, tnlCuda, int >& v2);
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< double, tnlCuda, int >& v1, const tnlVector< double, tnlCuda, int >& v2 );
//template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< long double, tnlCuda, int >& v1, const tnlVector< long double, tnlCuda, int >& v2 );
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< int, tnlCuda, long int >& v1, const tnlVector< int, tnlCuda, long int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< long int, tnlCuda, long int >& v1, const tnlVector< long int, tnlCuda, long int >& v2 );
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< float, tnlCuda, long int >& v1, const tnlVector< float, tnlCuda, long int >& v2 );
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< double, tnlCuda, long int >& v1, const tnlVector< double, tnlCuda, long int >& v2 );
//template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< long double, tnlCuda, long int >& v1, const tnlVector< long double, tnlCuda, long int >& v2 );

/****
 * Difference abs max
 */
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< int, tnlCuda, int >& v1, const tnlVector< int, tnlCuda, int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< long int, tnlCuda, int >& v1, const tnlVector< long int, tnlCuda, int >& v2 );
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< float, tnlCuda, int >& v1,  const tnlVector< float, tnlCuda, int >& v2);
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< double, tnlCuda, int >& v1, const tnlVector< double, tnlCuda, int >& v2 );
//template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< long double, tnlCuda, int >& v1, const tnlVector< long double, tnlCuda, int >& v2 );
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< int, tnlCuda, long int >& v1, const tnlVector< int, tnlCuda, long int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< long int, tnlCuda, long int >& v1, const tnlVector< long int, tnlCuda, long int >& v2 );
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< float, tnlCuda, long int >& v1, const tnlVector< float, tnlCuda, long int >& v2 );
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< double, tnlCuda, long int >& v1, const tnlVector< double, tnlCuda, long int >& v2 );
//template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< long double, tnlCuda, long int >& v1, const tnlVector< long double, tnlCuda, long int >& v2 );

/****
 * Difference abs min
 */
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< int, tnlCuda, int >& v1, const tnlVector< int, tnlCuda, int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< long int, tnlCuda, int >& v1, const tnlVector< long int, tnlCuda, int >& v2 );
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< float, tnlCuda, int >& v1,  const tnlVector< float, tnlCuda, int >& v2);
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< double, tnlCuda, int >& v1, const tnlVector< double, tnlCuda, int >& v2 );
//template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< long double, tnlCuda, int >& v1, const tnlVector< long double, tnlCuda, int >& v2 );
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< int, tnlCuda, long int >& v1, const tnlVector< int, tnlCuda, long int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< long int, tnlCuda, long int >& v1, const tnlVector< long int, tnlCuda, long int >& v2 );
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< float, tnlCuda, long int >& v1, const tnlVector< float, tnlCuda, long int >& v2 );
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< double, tnlCuda, long int >& v1, const tnlVector< double, tnlCuda, long int >& v2 );
//template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< long double, tnlCuda, long int >& v1, const tnlVector< long double, tnlCuda, long int >& v2 );
        
#endif
 