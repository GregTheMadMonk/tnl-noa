#include "hip/hip_runtime.h"
/***************************************************************************
                          tnlVectorOperationsCuda_impl.cu  -  description
                             -------------------
    begin                : Jul 20, 2013
    copyright            : (C) 2013 by Tomas Oberhuber
    email                : tomas.oberhuber@fjfi.cvut.cz
 ***************************************************************************/

/* See Copyright Notice in tnl/Copyright */

#include <core/vectors/tnlVectorOperations.h>

namespace TNL {

#ifdef TEMPLATE_EXPLICIT_INSTANTIATION

/****
 * Max
 */
template int         tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< long int, tnlCuda, int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< float, tnlCuda, int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< double, tnlCuda, int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< long double, tnlCuda, int >& v );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< long int, tnlCuda, long int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< float, tnlCuda, long int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< double, tnlCuda, long int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorMax( const tnlVector< long double, tnlCuda, long int >& v );
#endif
#endif

/****
 * Min
 */
template int         tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< long int, tnlCuda, int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< float, tnlCuda, int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< double, tnlCuda, int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< long double, tnlCuda, int >& v );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< long int, tnlCuda, long int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< float, tnlCuda, long int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< double, tnlCuda, long int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorMin( const tnlVector< long double, tnlCuda, long int >& v );
#endif
#endif

/****
 * Abs max
 */
template int         tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< long int, tnlCuda, int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< float, tnlCuda, int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< double, tnlCuda, int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< long double, tnlCuda, int >& v );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< long int, tnlCuda, long int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< float, tnlCuda, long int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< double, tnlCuda, long int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorAbsMax( const tnlVector< long double, tnlCuda, long int >& v );
#endif
#endif


/****
 * Abs min
 */
template int         tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< long int, tnlCuda, int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< float, tnlCuda, int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< double, tnlCuda, int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< long double, tnlCuda, int >& v );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< long int, tnlCuda, long int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< float, tnlCuda, long int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< double, tnlCuda, long int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorAbsMin( const tnlVector< long double, tnlCuda, long int >& v );
#endif
#endif

/****
 * L2 norm
 */
template int         tnlVectorOperations< tnlCuda >::getVectorL2Norm( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorL2Norm( const tnlVector< long int, tnlCuda, int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorL2Norm( const tnlVector< float, tnlCuda, int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorL2Norm( const tnlVector< double, tnlCuda, int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorL2Norm( const tnlVector< long double, tnlCuda, int >& v );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorL2Norm( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorL2Norm( const tnlVector< long int, tnlCuda, long int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorL2Norm( const tnlVector< float, tnlCuda, long int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorL2Norm( const tnlVector< double, tnlCuda, long int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorL2Norm( const tnlVector< long double, tnlCuda, long int >& v );
#endif
#endif

/****
 * L1 norm
 */
template int         tnlVectorOperations< tnlCuda >::getVectorL1Norm( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorL1Norm( const tnlVector< long int, tnlCuda, int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorL1Norm( const tnlVector< float, tnlCuda, int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorL1Norm( const tnlVector< double, tnlCuda, int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorL1Norm( const tnlVector< long double, tnlCuda, int >& v );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorL1Norm( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorL1Norm( const tnlVector< long int, tnlCuda, long int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorL1Norm( const tnlVector< float, tnlCuda, long int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorL1Norm( const tnlVector< double, tnlCuda, long int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorL1Norm( const tnlVector< long double, tnlCuda, long int >& v );
#endif
#endif

/****
 * Lp norm
 */
template int         tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< int, tnlCuda, int >& v, const int& p );
template long int    tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< long int, tnlCuda, int >& v, const long int& p );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< float, tnlCuda, int >& v, const float& p );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< double, tnlCuda, int >& v, const double& p );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< long double, tnlCuda, int >& v, const long double& p );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< int, tnlCuda, long int >& v, const int& p );
template long int    tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< long int, tnlCuda, long int >& v, const long int& p );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< float, tnlCuda, long int >& v, const float& p );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< double, tnlCuda, long int >& v, const double& p );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorLpNorm( const tnlVector< long double, tnlCuda, long int >& v, const long double& p );
#endif
#endif



/****
 * Sum
 */
template int         tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< int, tnlCuda, int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< long int, tnlCuda, int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< float, tnlCuda, int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< double, tnlCuda, int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< long double, tnlCuda, int >& v );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< int, tnlCuda, long int >& v );
template long int    tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< long int, tnlCuda, long int >& v );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< float, tnlCuda, long int >& v );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< double, tnlCuda, long int >& v );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorSum( const tnlVector< long double, tnlCuda, long int >& v );
#endif
#endif

/****
 * Difference max
 */
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< int, tnlCuda, int >& v1, const tnlVector< int, tnlCuda, int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< long int, tnlCuda, int >& v1, const tnlVector< long int, tnlCuda, int >& v2 );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< float, tnlCuda, int >& v1,  const tnlVector< float, tnlCuda, int >& v2);
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< double, tnlCuda, int >& v1, const tnlVector< double, tnlCuda, int >& v2 );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< long double, tnlCuda, int >& v1, const tnlVector< long double, tnlCuda, int >& v2 );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< int, tnlCuda, long int >& v1, const tnlVector< int, tnlCuda, long int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< long int, tnlCuda, long int >& v1, const tnlVector< long int, tnlCuda, long int >& v2 );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< float, tnlCuda, long int >& v1, const tnlVector< float, tnlCuda, long int >& v2 );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< double, tnlCuda, long int >& v1, const tnlVector< double, tnlCuda, long int >& v2 );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceMax( const tnlVector< long double, tnlCuda, long int >& v1, const tnlVector< long double, tnlCuda, long int >& v2 );
#endif
#endif

/****
 * Difference min
 */
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< int, tnlCuda, int >& v1, const tnlVector< int, tnlCuda, int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< long int, tnlCuda, int >& v1, const tnlVector< long int, tnlCuda, int >& v2 );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< float, tnlCuda, int >& v1,  const tnlVector< float, tnlCuda, int >& v2);
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< double, tnlCuda, int >& v1, const tnlVector< double, tnlCuda, int >& v2 );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< long double, tnlCuda, int >& v1, const tnlVector< long double, tnlCuda, int >& v2 );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< int, tnlCuda, long int >& v1, const tnlVector< int, tnlCuda, long int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< long int, tnlCuda, long int >& v1, const tnlVector< long int, tnlCuda, long int >& v2 );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< float, tnlCuda, long int >& v1, const tnlVector< float, tnlCuda, long int >& v2 );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< double, tnlCuda, long int >& v1, const tnlVector< double, tnlCuda, long int >& v2 );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceMin( const tnlVector< long double, tnlCuda, long int >& v1, const tnlVector< long double, tnlCuda, long int >& v2 );
#endif
#endif

/****
 * Difference abs max
 */
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< int, tnlCuda, int >& v1, const tnlVector< int, tnlCuda, int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< long int, tnlCuda, int >& v1, const tnlVector< long int, tnlCuda, int >& v2 );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< float, tnlCuda, int >& v1,  const tnlVector< float, tnlCuda, int >& v2);
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< double, tnlCuda, int >& v1, const tnlVector< double, tnlCuda, int >& v2 );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< long double, tnlCuda, int >& v1, const tnlVector< long double, tnlCuda, int >& v2 );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< int, tnlCuda, long int >& v1, const tnlVector< int, tnlCuda, long int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< long int, tnlCuda, long int >& v1, const tnlVector< long int, tnlCuda, long int >& v2 );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< float, tnlCuda, long int >& v1, const tnlVector< float, tnlCuda, long int >& v2 );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< double, tnlCuda, long int >& v1, const tnlVector< double, tnlCuda, long int >& v2 );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMax( const tnlVector< long double, tnlCuda, long int >& v1, const tnlVector< long double, tnlCuda, long int >& v2 );
#endif
#endif


/****
 * Difference abs min
 */
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< int, tnlCuda, int >& v1, const tnlVector< int, tnlCuda, int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< long int, tnlCuda, int >& v1, const tnlVector< long int, tnlCuda, int >& v2 );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< float, tnlCuda, int >& v1,  const tnlVector< float, tnlCuda, int >& v2);
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< double, tnlCuda, int >& v1, const tnlVector< double, tnlCuda, int >& v2 );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< long double, tnlCuda, int >& v1, const tnlVector< long double, tnlCuda, int >& v2 );
#endif

#ifdef INSTANTIATE_LONG_INT
template int         tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< int, tnlCuda, long int >& v1, const tnlVector< int, tnlCuda, long int >& v2 );
template long int    tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< long int, tnlCuda, long int >& v1, const tnlVector< long int, tnlCuda, long int >& v2 );
#ifdef INSTANTIATE_FLOAT
template float       tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< float, tnlCuda, long int >& v1, const tnlVector< float, tnlCuda, long int >& v2 );
#endif
template double      tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< double, tnlCuda, long int >& v1, const tnlVector< double, tnlCuda, long int >& v2 );
#ifdef INSTANTIATE_LONG_DOUBLE
template long double tnlVectorOperations< tnlCuda >::getVectorDifferenceAbsMin( const tnlVector< long double, tnlCuda, long int >& v1, const tnlVector< long double, tnlCuda, long int >& v2 );
#endif
#endif
 
#endif
 
} // namespace TNL