#include "hip/hip_runtime.h"
/***************************************************************************
                          cuda-reduction-diff-max_impl.cu  -  description
                             -------------------
    begin                : Jan 19, 2014
    copyright            : (C) 2014 by Tomas Oberhuber
    email                : tomas.oberhuber@fjfi.cvut.cz
 ***************************************************************************/

/* See Copyright Notice in tnl/Copyright */
 
#include <core/cuda/reduction-operations.h>
#include <core/cuda/cuda-reduction.h>
 
#ifdef TEMPLATE_EXPLICIT_INSTANTIATION

/****
 * Diff max
 */

template bool reductionOnCudaDevice< tnlParallelReductionDiffMax< char, int > >
                                   ( tnlParallelReductionDiffMax< char, int >& operation,
                                     const typename tnlParallelReductionDiffMax< char, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffMax< char, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffMax< char, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffMax< char, int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffMax< int, int > >
                                   ( tnlParallelReductionDiffMax< int, int >& operation,
                                     const typename tnlParallelReductionDiffMax< int, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffMax< int, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffMax< int, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffMax< int, int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffMax< float, int > >
                                   ( tnlParallelReductionDiffMax< float, int >& operation,
                                     const typename tnlParallelReductionDiffMax< float, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffMax< float, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffMax< float, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffMax< float, int> :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffMax< double, int > >
                                   ( tnlParallelReductionDiffMax< double, int>& operation,
                                     const typename tnlParallelReductionDiffMax< double, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffMax< double, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffMax< double, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffMax< double, int> :: ResultType& result );

#ifdef INSTANTIATE_LONG_DOUBLE
template bool reductionOnCudaDevice< tnlParallelReductionDiffMax< long double, int > >
                                   ( tnlParallelReductionDiffMax< long double, int>& operation,
                                     const typename tnlParallelReductionDiffMax< long double, int > :: IndexType size,
                                     const typename tnlParallelReductionDiffMax< long double, int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffMax< long double, int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffMax< long double, int> :: ResultType& result );
#endif

#ifdef INSTANTIATE_LONG_INT
template bool reductionOnCudaDevice< tnlParallelReductionDiffMax< char, long int > >
                                   ( tnlParallelReductionDiffMax< char, long int >& operation,
                                     const typename tnlParallelReductionDiffMax< char, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffMax< char, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffMax< char, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffMax< char, long int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffMax< int, long int > >
                                   ( tnlParallelReductionDiffMax< int, long int >& operation,
                                     const typename tnlParallelReductionDiffMax< int, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffMax< int, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffMax< int, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffMax< int, long int > :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffMax< float, long int > >
                                   ( tnlParallelReductionDiffMax< float, long int >& operation,
                                     const typename tnlParallelReductionDiffMax< float, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffMax< float, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffMax< float, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffMax< float, long int> :: ResultType& result );

template bool reductionOnCudaDevice< tnlParallelReductionDiffMax< double, long int > >
                                   ( tnlParallelReductionDiffMax< double, long int>& operation,
                                     const typename tnlParallelReductionDiffMax< double, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffMax< double, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffMax< double, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffMax< double, long int> :: ResultType& result );

#ifdef INSTANTIATE_LONG_DOUBLE
template bool reductionOnCudaDevice< tnlParallelReductionDiffMax< long double, long int > >
                                   ( tnlParallelReductionDiffMax< long double, long int>& operation,
                                     const typename tnlParallelReductionDiffMax< long double, long int > :: IndexType size,
                                     const typename tnlParallelReductionDiffMax< long double, long int > :: RealType* deviceInput1,
                                     const typename tnlParallelReductionDiffMax< long double, long int > :: RealType* deviceInput2,
                                     typename tnlParallelReductionDiffMax< long double, long int> :: ResultType& result );
#endif
#endif
#endif
