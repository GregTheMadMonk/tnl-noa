#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <iomanip>

#include <TNL/Containers/Array.h>

#include "../../src/quicksort_dynamic/quicksort.cuh"
#include "../../src/util/timer.h"

using namespace TNL;
using namespace TNL::Containers;
using namespace std;

typedef Devices::Cuda Device;

int main()
{
    srand(8151);
    for(int pow = 5; pow <= 23; pow++)
    {
        int size =(1<< pow);

        vector<int> vec(size);
        iota(vec.begin(), vec.end(), 0);

        Array<int, Device> arr;
        vector<double> resAcc;

        //sorted sequence
        {
            arr = vec;
            auto view = arr.getView();

            {
                TIMER t([&](double res){resAcc.push_back(res);});
                quicksort(view);
            }
        }

        //almost sorted sequence
        {
            for(int i = 0; i < 3; i++)
            {
                int s = rand() % (size - 3);
                std::swap(vec[s], vec[s + 1]);
            }

            arr = vec;
            auto view = arr.getView();

            {
                TIMER t([&](double res){resAcc.push_back(res);});
                quicksort(view);
            }
        }

        //decreasing sequence
        {
            for(size_t i = 0; i < size; i++)
                vec[i] = -i;
                
            arr = vec;
            auto view = arr.getView();

            {
                TIMER t([&](double res){resAcc.push_back(res);});
                quicksort(view);
            }
        }
        
        //random sequence
        {
            random_shuffle(vec.begin(), vec.end());

            arr = vec;
            auto view = arr.getView();

            {
                TIMER t([&](double res){resAcc.push_back(res);});
                quicksort(view);
            }
        }


        cout << "2^" << pow << " = ";
        cout << fixed;
        cout << setprecision(3);
        cout << (accumulate(resAcc.begin(), resAcc.end(), 0.0) / resAcc.size()) << " ms" << endl;
    }

    return 0;
}